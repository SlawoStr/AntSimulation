#include "hip/hip_runtime.h"
#include "AntManager.cuh"

void AntManager::setSettings(float maxSpeed, float maxForce, float wanderDistance, float wanderRadius, float2 simulationBound, unsigned agentNumber)
{
	// Set settings
	m_maxSpeed = maxSpeed;
	m_maxForce = maxForce;
	m_wanderDistance = wanderDistance;
	m_wanderRadius = wanderRadius;
	m_simulationBound = simulationBound;
	// Allocate memory for agents data
	m_positions.reserve(agentNumber);
	m_velocity.reserve(agentNumber);
	m_acceleration.reserve(agentNumber);
	m_angle.reserve(agentNumber);
}

void AntManager::addAnt(float2 positions, float angle)
{
	m_positions.push_back(positions);
	m_velocity.push_back({ 0.f,0.f });
	m_acceleration.push_back({ 0.f,0.f });
	m_angle.push_back(angle);
}